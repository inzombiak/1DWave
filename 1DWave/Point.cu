#include "hip/hip_runtime.h"
#include "Point.h"

void DiskParameters::SetParameters()
{
	BIG_TWO = 2;
	deltaTau = deltaRo / Us0;
	chi = double(0.5) / (Us0*Us0);
	gammaSq = (chi / BIG_TWO) - 1;
	alpha = (chi / BIG_TWO) * (double(0.5) - chi / BIG_TWO);
	C1 = deltaTau * alpha / BIG_TWO;
	C2 = deltaTau * gammaSq / (BIG_TWO * BIG_TWO);
	C3 = C1 / Us0;
	C4 = C2 / Us0;
}

Point::Point() : ro(0), tau(0), dSigmaDRo(0), dSigmaDTau(0), sigma(0){};

bool Point::SetFromFile(std::ifstream& file, bool deltaSigmaDeltaTauFlag)
{
	if (!(file >> ro))
		return false;
	file >> sigma;
	file >> dSigmaDRo;

	if (deltaSigmaDeltaTauFlag)
		dSigmaDTau = 0;
	else
		dSigmaDTau = sigma*sigma;
	tau = 0;

	return true;
}

void Point::Set(const Point& left, const Point& right, const DiskParameters& params)
{
	double lastSigma;

	tau = left.tau + params.deltaTau;
	ro = right.ro - params.deltaRo;
	CalcDSigmaDRoSecondApprox(left, right, params);
	CalcDSigmaDTau(left, right, params);
	CalcSigma(left, *this, params);
	lastSigma = 0;

	while (abs(sigma - lastSigma) > params.Epsilon)
	{
		lastSigma = sigma;
		CalcDSigmaDTauSecondApprox(left, right, *this, params);
		CalcSigma(left, *this, params);
	}

}

void Point::Extrapolate(const Point& p1, const Point& p2, const DiskParameters& params)
{
	ro = p1.ro + params.deltaRo;
	tau = p1.tau;
	sigma = 0;
	dSigmaDRo = p1.dSigmaDRo;
	dSigmaDTau = p1.dSigmaDTau;
}

void Point::CalcDSigmaDTau(const Point& left, const Point& right, const DiskParameters& params)
{
	double leftPow, rightPow, result = 0;
	leftPow = pow(left.sigma, 3);
	rightPow = pow(right.sigma, 3);

	result += params.Us0*(left.dSigmaDTau + right.dSigmaDTau);
	result += -(params.Us0*params.Us0)*(left.dSigmaDRo - right.dSigmaDRo);
	result += -params.deltaRo*(params.BIG_TWO * params.alpha*(leftPow + rightPow) - params.gammaSq*(left.sigma + right.sigma));
	result /= (params.BIG_TWO*params.Us0);

	/*result += left.dSigmaDTau + right.dSigmaDTau;
	result += params.Us0*(right.dSigmaDRo-left.dSigmaDRo);
	result /= BIG_TWO;
	result -= BIG_TWO * C1 * (leftPow + rightPow);
	result += BIG_TWO * C2 * (left.sigma + right.sigma);*/

	if (abs(result) < pow((double)10, (double)-6))
		result = 0;
	dSigmaDTau = result;
}

void Point::CalcDSigmaDTauSecondApprox(const Point& left, const Point& right, const Point& newPoint, const DiskParameters& params)
{
	double leftPow, rightPow, newPow, result = 0;
	leftPow = pow(left.sigma, 3);
	rightPow = pow(right.sigma, 3);
	newPow = pow(newPoint.sigma, 3);

	result += params.Us0*(left.dSigmaDTau + right.dSigmaDTau);
	result += (params.Us0*params.Us0)*(right.dSigmaDRo - left.dSigmaDRo);
	result += -params.deltaRo*params.alpha*(params.BIG_TWO * newPow + leftPow + rightPow);
	result += (params.deltaRo / 2)* (-params.gammaSq) * (params.BIG_TWO * newPoint.sigma + left.sigma + right.sigma);
	result /= (params.BIG_TWO * params.Us0);

	/*result += left.dSigmaDTau + right.dSigmaDTau;
	result += params.Us0*(right.dSigmaDRo - left.dSigmaDRo);
	result /= BIG_TWO;
	result -= C1 * (leftPow + rightPow + BIG_TWO*newPow);
	result += C2 * (left.sigma + right.sigma + newPoint.sigma);*/

	if (abs(result) < pow((double)10, (double)-6))
		result = 0;
	dSigmaDTau = result;
}

void Point::CalcDSigmaDRoSecondApprox(const Point& left, const Point& right, const DiskParameters& params)
{
	double leftPow, rightPow, result = 0;
	leftPow = left.sigma;
	rightPow = right.sigma;
	leftPow = pow(left.sigma, 3);
	rightPow = pow(right.sigma, 3);

	result += (params.Us0*params.Us0)*(right.dSigmaDRo + left.dSigmaDRo);
	result += params.Us0*(right.dSigmaDTau - left.dSigmaDTau);
	result += params.deltaRo*(params.BIG_TWO * params.alpha*(leftPow - rightPow) - params.gammaSq*(left.sigma - right.sigma)) / 2;
	result /= (params.BIG_TWO* params.Us0*params.Us0);

	/*result += right.dSigmaDRo + left.dSigmaDRo;
	result +=(right.dSigmaDTau - left.dSigmaDTau) / params.Us0;
	result /= BIG_TWO;
	result -= C3*(rightPow - leftPow);
	result += C4*(right.sigma - left.sigma);*/

	if (abs(result) < pow((double)10, (double)-6))
		result = 0;
	dSigmaDRo = result;
}

void Point::CalcSigma(const Point& left, const Point& right, const DiskParameters& params)
{
	double result = 0;

	result += (left.dSigmaDTau + right.dSigmaDTau)*params.deltaTau / 2;
	result += (left.dSigmaDRo + right.dSigmaDRo)*params.deltaRo / 2;
	result += left.sigma;

	if (abs(result) < pow((double)10, (double)-6))
		result = 0;
	sigma = result;
}

void Point::Barrier(const Point& copy)
{
	sigma = copy.sigma;
	tau = copy.tau;
	dSigmaDRo = -copy.dSigmaDRo;
	dSigmaDTau = copy.dSigmaDTau;

}